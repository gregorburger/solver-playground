#include "solve_cusparse.h"
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
using namespace std;

//CudasparseSafeCall
#define CSC(func) { \
    hipsparseStatus_t status = func; \
    if (status != HIPSPARSE_STATUS_SUCCESS) { \
        cout << "error calling: " << #func << endl; \
    } \
}

#define CUDA_SC(func) { \
    hipError_t error; \
    error = func; \
    if (error != hipSuccess) {\
        cout << "error calling: " << #func << endl; \
    }\
}

void solve_cusparse(csr_matrix &matrix, const char *finput) {
    hipsparseHandle_t handle;
    CSC(hipsparseCreate(&handle));

    hipsparseMatDescr_t desc;
    CSC(hipsparseCreateMatDescr(&desc));

    CSC(hipsparseSetMatDiagType(desc, HIPSPARSE_DIAG_TYPE_NON_UNIT));
    CSC(hipsparseSetMatFillMode(desc, HIPSPARSE_FILL_MODE_LOWER))
    CSC(hipsparseSetMatIndexBase(desc, HIPSPARSE_INDEX_BASE_ONE))
    CSC(hipsparseSetMatType(desc, HIPSPARSE_MATRIX_TYPE_TRIANGULAR));

    double *values, *x, *y;
    int *row_index, *columns;

    CUDA_SC(hipMalloc(&values,     sizeof(double) *    matrix.nnz));
    CUDA_SC(hipMalloc(&x,          sizeof(double) *    matrix.n));
    CUDA_SC(hipMalloc(&y,          sizeof(double) *    matrix.n));
    CUDA_SC(hipMalloc(&row_index,  sizeof(int) *       (matrix.n+1)));
    CUDA_SC(hipMalloc(&columns,    sizeof(int) *       matrix.nnz));


    CUDA_SC(hipMemcpy(values, matrix.values,       matrix.nnz * sizeof(double),    hipMemcpyHostToDevice));
    CUDA_SC(hipMemcpy(x, matrix.b,                 matrix.n * sizeof(double),    hipMemcpyHostToDevice));
    CUDA_SC(hipMemcpy(columns, matrix.columns,     matrix.nnz * sizeof(int),       hipMemcpyHostToDevice));
    CUDA_SC(hipMemcpy(row_index, matrix.row_index, (matrix.n+1) * sizeof(int),     hipMemcpyHostToDevice));


    cusparseSolveAnalysisInfo_t info;
    cusparseCreateSolveAnalysisInfo(&info);

    hipsparseStatus_t status;

    status = cusparseDcsrsv_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, matrix.n, matrix.nnz, desc, values, row_index, columns, info);
    assert(status == HIPSPARSE_STATUS_SUCCESS);

    double alpha = 1.0;

    status = cusparseDcsrsv_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, matrix.n, &alpha, desc, values, row_index, columns, info, x, y);
    assert(status == HIPSPARSE_STATUS_SUCCESS);

    status = cusparseDcsrsv_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, matrix.n, &alpha, desc, values, row_index, columns, info, x, y);
    assert(status == HIPSPARSE_STATUS_SUCCESS);

    status = cusparseDcsrsv_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, matrix.n, &alpha, desc, values, row_index, columns, info, x, y);
    assert(status == HIPSPARSE_STATUS_SUCCESS);

    status = cusparseDcsrsv_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, matrix.n, &alpha, desc, values, row_index, columns, info, x, y);
    assert(status == HIPSPARSE_STATUS_SUCCESS);

    double *host_y = new double[matrix.n];

    CUDA_SC(hipMemcpy(host_y, y, sizeof(double) * matrix.n, hipMemcpyDeviceToHost));

    for (int i = 0; i < matrix.n; i++) {
        printf("%f %f\n", host_y[i], matrix.x[i]);
    }

    //clean up
    cusparseDestroySolveAnalysisInfo(info);

    CUDA_SC(hipFree(columns));
    CUDA_SC(hipFree(row_index));
    CUDA_SC(hipFree(y));
    CUDA_SC(hipFree(x));
    CUDA_SC(hipFree(values));

    CSC(hipsparseDestroyMatDescr(desc));

    CSC(hipsparseDestroy(handle));
}
